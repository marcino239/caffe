
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNSoftmaxOldLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  CUDNN_CHECK(cudnnSoftmaxOldForward(handle_, CUDNN_SOFTMAXOLD_ACCURATE,
      CUDNN_SOFTMAXOLD_MODE_CHANNEL,
      bottom_desc_, bottom_data, top_desc_, top_data));
}

template <typename Dtype>
void CuDNNSoftmaxOldLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    CUDNN_CHECK(cudnnSoftmaxOldBackward(handle_, CUDNN_SOFTMAXOLD_ACCURATE,
        CUDNN_SOFTMAXOLD_MODE_CHANNEL,
        top_desc_, top_data, top_desc_, top_diff, bottom_desc_, bottom_diff));
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNSoftmaxOldLayer);

}  // namespace caffe
#endif
